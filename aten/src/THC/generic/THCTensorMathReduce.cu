
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathReduce.cu"
#else

THC_API void
THCTensor_(sum)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<real>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(0),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(prod)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<real>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(1),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(mean)(THCState *state, THCTensor *self, THCTensor *src, int dim, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  const accreal size = scalar_cast<accreal>(THCTensor_(size)(state, src, dim));
  if (!THC_reduceDim<real>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           ReduceDivide<accreal>{size},
                           scalar_cast<accreal>(0),
                           dim,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

THC_API void
THCTensor_(renorm)(THCState *state, THCTensor* self, THCTensor* src, real value, int dimension, real maxnorm)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  THCTensor *self_;
  THCTensor *src_ = THCTensor_(newTranspose)(state, src, dimension, 0);
  THCTensor *data = THCTensor_(newClone)(state, src_);
  int64_t numel = THCTensor_(nElement)(state, data);

  THArgCheck(dimension >= 0 && dimension < THCTensor_(nDimensionLegacyNoScalars)(state, src), 3, "invalid dimension");
  THArgCheck(THCNumerics<real>::gt(value, scalar_cast<real>(0)), 2, "non-positive-norm not supported");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) > 1, 1, "need at least 2 dimensions");

  if (numel > 0) {
    ptrdiff_t size = numel / THTensor_sizeLegacyNoScalars(data, 0);
    dim3 grid( THTensor_sizeLegacyNoScalars(data, 0));
    dim3 threads(32);

    THCTensor_kernel_renorm<real, accreal>
      <<<grid, threads, 0, THCState_getCurrentStream(state)>>>
      (THCTensor_(data)(state, data), scalar_cast<accreal>(value), size, scalar_cast<accreal>(maxnorm));

    cudaError errcode = cudaGetLastError();
    if(errcode != cudaSuccess)
      THError(cudaGetErrorString(errcode));
  }

  THCTensor_(free)(state, src_);
  self_ = THCTensor_(newTranspose)(state, data, dimension, 0);
  THCTensor_(resizeAs)(state, self, self_);
  THCTensor_(freeCopyTo)(state, self_, self);
  THCTensor_(free)(state, data);
}

THC_API void
THCTensor_(std)(THCState *state, THCTensor *self_, THCTensor *src, int dimension, int biased, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));

  THCTensor_preserveReduceDimSemantics(
      state, self_, THCTensor_(nDimensionLegacyAll)(state, src), dimension, keepdim);
  std::vector<int64_t> dim = THTensor_sizesLegacyNoScalars(src);
  dim[dimension] = 1;
  THCTensor_(resize)(state, self_, dim, {});

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  src = THCTensor_(newContiguous)(state, src);

  if (dimension == THCTensor_(nDimensionLegacyAll)(state, src) - 1) {
    THCTensor_varInnermostDim<THCTensor, real, accreal, true>(state, self, src, biased);
  } else {
    THCTensor_varOuterDim<THCTensor, real, accreal, true>(state, self, src, dimension, biased);
  }

  THCTensor_(free)(state, src);
  THCTensor_(freeCopyTo)(state, self, self_);

  if (!keepdim) {
    THCTensor_(squeeze1d)(state, self_, self_, dimension);
  }
}

THC_API void
THCTensor_(var)(THCState *state, THCTensor *self_, THCTensor *src, int dimension, int biased, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));

  THCTensor_preserveReduceDimSemantics(
      state, self_, THCTensor_(nDimensionLegacyAll)(state, src), dimension, keepdim);
  std::vector<int64_t> dim = THTensor_sizesLegacyNoScalars(src);
  dim[dimension] = 1;
  THCTensor_(resize)(state, self_, dim, {});

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  src = THCTensor_(newContiguous)(state, src);

  if (dimension == THCTensor_(nDimensionLegacyAll)(state, src) - 1) {
    THCTensor_varInnermostDim<THCTensor, real, accreal, false>(state, self, src, biased);
  } else {
    THCTensor_varOuterDim<THCTensor, real, accreal, false>(state, self, src, dimension, biased);
  }

  THCTensor_(free)(state, src);
  THCTensor_(freeCopyTo)(state, self, self_);

  if (!keepdim) {
    THCTensor_(squeeze1d)(state, self_, self_, dimension);
  }
}

THC_API accreal
THCTensor_(stdall)(THCState *state, THCTensor *self, int biased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCNumerics<accreal>::sqrt((THCTensor_(varall)(state, self, biased)));
}

THC_API accreal
THCTensor_(varall)(THCState *state, THCTensor *self, int biased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal mean = THCTensor_(meanall)(state, self);

  accreal val;
  if (!THC_reduceAll<real>(state, self,
                           SquareFunctor<accreal>(mean),
                           ReduceAdd<accreal>(),
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  val = THCNumerics<accreal>::div(
    val,
    scalar_cast<accreal>(std::max<int64_t>(0, THCTensor_(nElement)(state, self) - (biased ? 0 : 1)))
  );

  THCudaCheck(cudaGetLastError());
  return val;
}

THC_API void
THCTensor_(norm)(THCState *state, THCTensor* self, THCTensor* src, real _value, int dimension, int keepdim)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    THC_reduceDim<real>(state, self, src,
                        TensorNonZeroOp<accreal>{},
                        ReduceAdd<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(1))) {
    THC_reduceDim<real>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceAdd<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(2))) {
    THC_reduceDim<real>(state, self, src,
                        TensorNormOp<accreal, 2>{value},
                        ReduceAdd<accreal>{},
                        ReducePow<accreal>{scalar_cast<accreal>(.5)},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    THC_reduceDim<real>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMax<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else {
    THC_reduceDim<real>(state, self, src,
                        TensorNormOp<accreal, -1>{value},
                        ReduceAdd<accreal>{},
                        ReducePow<accreal>{THCNumerics<accreal>::cinv(value)},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API accreal
THCTensor_(normall)(THCState *state, THCTensor *self, real _value)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal result;

  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    THC_reduceAll<real>(state, self,
                        TensorNonZeroOp<accreal>{},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(1))) {
    THC_reduceAll<real>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(2))) {
    THC_reduceAll<real>(state, self,
                        TensorNormOp<accreal, 2>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
    result = THCNumerics<accreal>::sqrt(result);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    THC_reduceAll<real>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMax<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else {
    THC_reduceAll<real>(state, self,
                        TensorNormOp<accreal, -1>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
    result = THCNumerics<accreal>::pow(result, 
                                       THCNumerics<accreal>::cinv(value));
  }

  THCudaCheck(cudaGetLastError());
  return result;
}

accreal THCTensor_(dist)(THCState *state, THCTensor *self,
                         THCTensor *src, real _value)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  self = THCTensor_(newContiguous)(state, self);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  src = THCTensor_(newContiguous)(state, src);
  thrust::device_ptr<real> self_data(THCTensor_(data)(state, self));
  thrust::device_ptr<real> src_data(THCTensor_(data)(state, src));

  THCThrustAllocator thrustAlloc(state);
  accreal result = thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+size, src_data, scalar_cast<accreal>(0),
    thrust::plus<accreal>(),
    ThrustTensorDistOp<real, accreal>(value));

  THCTensor_(free)(state, src);
  THCTensor_(free)(state, self);

  return THCNumerics<accreal>::pow(result, THCNumerics<accreal>::cinv(value));
}

#endif

THC_API accreal
THCTensor_(sumall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<real>(state, self,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return val;
}

THC_API accreal
THCTensor_(prodall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<real>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           scalar_cast<accreal>(1),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return val;
}

THC_API accreal
THCTensor_(meanall)(THCState *state, THCTensor *self)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCTensor_(sumall)(state, self)/THCTensor_(nElement)(state, self);
}

THC_API real
THCTensor_(minall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<real>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMin<accreal>{},
                           THCNumerics<accreal>::max(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<real>(val);
}

THC_API real
THCTensor_(maxall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<real>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMax<accreal>{},
                           THCNumerics<accreal>::min(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<real>(val);
}

THC_API real
THCTensor_(medianall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));

  real val;
  ptrdiff_t nelem, k;

  nelem = THCTensor_(nElement)(state, self);
  k = (nelem-1) >> 1;

  THCTensor *view = THCTensor_(newView)(state, self, {nelem});

  THCTensor *sorted = THCTensor_(new)(state);
  THCudaLongTensor *indices = THCudaLongTensor_new(state);

  THCTensor_(sort)(state, sorted, indices, view, 0, 0);

  val = THCTensor_(get1d)(state, sorted, k);

  THCTensor_(free)(state, view);
  THCTensor_(free)(state, sorted);
  THCudaLongTensor_free(state, indices);

  THCudaCheck(cudaGetLastError());

  return val;
}

THC_API void
THCTensor_(median)(THCState *state,
                   THCTensor *values,
                   THCudaLongTensor *indices,
                   THCTensor *self,
                   int dimension,
                   int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));

  int64_t t_size_dim, k;

  t_size_dim = THCTensor_(size)(state, self, dimension);

  k = (t_size_dim-1) >> 1;

  THCTensor *sorted = THCTensor_(new)(state);
  THCudaLongTensor *sorted_indices = THCudaLongTensor_new(state);

  THCTensor_(sort)(state, sorted, sorted_indices, self, dimension, 0);

  THCTensor *newValues = THCTensor_(newNarrow)(state, sorted, dimension, k, 1);
  THCudaLongTensor *newIndices = THCudaLongTensor_newNarrow(state, sorted_indices, dimension, k, 1);

  THCTensor_(free)(state, sorted);
  THCudaLongTensor_free(state, sorted_indices);

  if (!keepdim) {
    THCTensor_(squeeze1d)(state, newValues, newValues, dimension);
    THCudaLongTensor_squeeze1d(state, newIndices, newIndices, dimension);
  }

  THCTensor_(resizeAs)(state, values, newValues);
  THCudaLongTensor_resizeAs(state, indices, newIndices);
  THCTensor_(copy)(state, values, newValues);
  THCudaLongTensor_copy(state, indices, newIndices);

  THCTensor_(free)(state, newValues);
  THCudaLongTensor_free(state, newIndices);

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(max)(THCState *state,
                THCTensor *values,
                THCudaLongTensor *indices,
                THCTensor *src,
                int dimension,
                int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<real, int64_t>
    init =
    thrust::make_pair<real, int64_t>(
      THCNumerics<real>::min(), 0);

  return THC_reduceDimIndex<real, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MaxValuePair<real, int64_t>());
}

THC_API void
THCTensor_(min)(THCState *state,
                THCTensor *values,
                THCudaLongTensor *indices,
                THCTensor *src,
                int dimension,
                int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<real, int64_t>
    init =
    thrust::make_pair<real, int64_t>(
      THCNumerics<real>::max(), 0);

  return THC_reduceDimIndex<real, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MinValuePair<real, int64_t>());
}

#endif
